
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define N 1000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    for(int i = index; i < n; i += stride){
        out[i] = a[i] + b[i];
    }
}

int main(){
    hipProfilerStart();

    float *a, *b, *out; 

    // Allocate memory
    hipMallocManaged(&a, sizeof(float) * N);
    hipMallocManaged(&b, sizeof(float) * N);
    hipMallocManaged(&out, sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Main function
    vector_add<<<1,256>>>(out, a, b, N);

    hipDeviceSynchronize();

    hipFree(a);
    hipFree(b);
    hipFree(out);

    hipDeviceReset();
    hipProfilerStop();
}
