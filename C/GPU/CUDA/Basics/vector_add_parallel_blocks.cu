
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define N 1000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = index; i < n; i += stride){
        out[i] = a[i] + b[i];
    }
}

int main(){
    hipProfilerStart();

    float *a, *b, *out; 

    // Allocate memory
    hipMallocManaged(&a, sizeof(float) * N);
    hipMallocManaged(&b, sizeof(float) * N);
    hipMallocManaged(&out, sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Main function
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    vector_add<<<numBlocks, blockSize>>>(out, a, b, N);

    hipDeviceSynchronize();

    hipFree(a);
    hipFree(b);
    hipFree(out);

    hipDeviceReset();
    hipProfilerStop();
}
