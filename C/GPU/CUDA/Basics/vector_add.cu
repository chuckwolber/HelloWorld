
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define N 1000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    hipProfilerStart();

    float *a, *b, *out; 

    // Allocate memory
    hipMallocManaged(&a, sizeof(float) * N);
    hipMallocManaged(&b, sizeof(float) * N);
    hipMallocManaged(&out, sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Main function
    vector_add<<<1,1>>>(out, a, b, N);

    hipDeviceSynchronize();

    hipFree(a);
    hipFree(b);
    hipFree(out);

    hipDeviceReset();
    hipProfilerStop();
}
