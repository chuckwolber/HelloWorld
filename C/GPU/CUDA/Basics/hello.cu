
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    hipProfilerStart();

    cuda_hello<<<1,1>>>(); 

    hipDeviceSynchronize();
    hipDeviceReset();
    hipProfilerStop();

    return 0;
}

